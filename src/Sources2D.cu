#include "hip/hip_runtime.h"
#include <TH.h>
#include <utils.h>
#include <math.h>

THSinHz::THSinHz(THGrid * grid,int x,int y,double amplitude,double frequency,double phase)
{
    this->amplitude = amplitude;
    this->frequency = frequency;
    this->phase = phase;
    this->grid = grid;
    this->x = x;
    this->y = y;
    checkWaveStability(grid,frequency);
}

__global__ void cuda_update(THGrid * grid,int xy,double amplitude,double frequency, double phase)
{
    THCell * cells = grid->cells[grid->current];
    cells[xy].Hz = amplitude*sin(grid->time*frequency+phase);
}

void update(THSinHz * sinHz)
{
    THGrid * grid;

    int xy = XY_TO_INDEX(sinHz->x,sinHz->y,sinHz->grid->space.nx,sinHz->grid->space.ny);

    hipMalloc((void **)&grid,sizeof(THGrid));
    hipMemcpy(grid,sinHz->grid,sizeof(THGrid),hipMemcpyHostToDevice);

    cuda_update<<<1,1>>>(grid,xy,sinHz->amplitude,sinHz->frequency,sinHz->phase);
    hipFree(grid);
}
