#include "hip/hip_runtime.h"
#include <TH.h>
#include <utils.h>
#include <math.h>

THSinHz::THSinHz(THGrid * grid,int x,int y,double amplitude,double frequency,double phase)
{
    this->amplitude = amplitude;
    this->frequency = frequency;
    this->phase = phase;
    this->x = x;
    this->y = y;
}


__global__  void update(THGrid * grid,THSinHz * sinHz)
{
    THCell * cells = grid->cells[grid->current];
    int xy = XY_TO_INDEX(sinHz->x,sinHz->y,grid->space.nx,grid->space.ny);
    cells[xy].Hz = sinHz->amplitude*sin(grid->time*sinHz->frequency+ sinHz->phase)/mu_0;
}
