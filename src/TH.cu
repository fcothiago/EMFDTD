#include "hip/hip_runtime.h"
#include <TH.h>
#include <utils.h>
#include <iostream>

#define THREADS_PER_BLOCK 100

THGrid::THGrid(unsigned int nx, unsigned int ny, double dt, double ds):Grid2D(nx,ny,dt,ds)
{
    THCell * array = new THCell[nx*ny];

    hipError_t err1 = hipMalloc((void **)&this->cells[0],sizeof(THCell)*nx*ny);
    hipError_t err2 = hipMalloc((void **)&this->cells[1],sizeof(THCell)*nx*ny);

    if(err1 != hipSuccess)
        std::cout << hipGetErrorString(err1) << std::endl;
    if(err2 != hipSuccess)
        std::cout << hipGetErrorString(err2) << std::endl;

    err1 = hipMemcpy(this->cells[0],array,sizeof(THCell)*nx*ny,hipMemcpyHostToDevice);
    err2 = hipMemcpy(this->cells[1],array,sizeof(THCell)*nx*ny,hipMemcpyHostToDevice);

    if(err1 != hipSuccess)
        std::cout << hipGetErrorString(err1) << std::endl;
    if(err2 != hipSuccess)
        std::cout << hipGetErrorString(err2) << std::endl;

    delete array;
}

THGrid::~THGrid()
{
    hipFree((void **)&this->cells[0]);
    hipFree((void **)&this->cells[1]);
}

THCell::THCell()
{
    this->Ex = 0;
    this->Ey = 0;
    this->Hz = 0;
    this->Epsx = eps_0;
    this->Epsy = eps_0;
    this->Muz = mu_0;
    this->Rhoxe = 0;
    this->Rhoye = 0;
    this->Rhozm = 0;
}
 
__device__ void updateE(THGrid * grid,int & x,int & y)
{   
    const unsigned int lx = XY_TO_INDEX(x-1,y,grid->space.nx,grid->space.ny);
    const unsigned int ly = XY_TO_INDEX(x,y-1,grid->space.nx,grid->space.ny);
    const unsigned int xy = XY_TO_INDEX(x,y,grid->space.nx,grid->space.ny);

    THCell * next = & grid->cells[1-grid->current][xy] ;
    THCell * old = grid->cells[grid->current] ;

    const double Coef_eex = (2*next->Epsx-grid->space.dt*next->Rhoxe)
                    /(2*next->Epsx+grid->space.dt*next->Rhoxe);
    const double Coef_ehx = (2*grid->space.dt)
                    /((2*next->Epsx+grid->space.dt*next->Rhoxe)*grid->space.ds);
    const double Coef_eix = (-2*grid->space.dt)
                    /(2*next->Epsx+grid->space.dt*next->Rhoxe);

    const double Coef_eey = (2*next->Epsy-grid->space.dt*next->Rhoye)
                    /(2*next->Epsy+grid->space.dt*next->Rhoye);
    const double Coef_ehy = (- 2*grid->space.dt)
                    /((2*next->Epsy+grid->space.dt*next->Rhoye)*grid->space.ds);
    const double Coef_eiy = (-2*grid->space.dt)
                    /(2*next->Epsy+grid->space.dt*next->Rhoye);

    next->Ex = Coef_eex*old[xy].Ex
            + Coef_ehx*(old[xy].Hz - old[ly].Hz)
            + Coef_eix*(old[xy].Jx);


    next->Ey = Coef_eey*old[xy].Ey 
            + Coef_ehy*(old[xy].Hz - old[lx].Hz) 
            + Coef_eiy*(old[xy].Jy);
}

__device__ void updateH(THGrid * grid,int & x,int & y)
{
    const long unsigned int ux = XY_TO_INDEX(x+1,y,grid->space.nx,grid->space.ny);
    const long unsigned int uy = XY_TO_INDEX(x,y+1,grid->space.nx,grid->space.ny);
    const long unsigned int xy = XY_TO_INDEX(x,y,grid->space.nx,grid->space.ny);

    THCell * next = & grid->cells[1-grid->current][xy] ;
    THCell * old = grid->cells[grid->current] ;

    const double Coef_hhz = (2*next->Muz- grid->space.dt*next->Rhozm)
                            /(2*next->Muz+ grid->space.dt*next->Rhozm);

    const double Coef_hex = (2*grid->space.dt)
                            /((2*next->Muz+ grid->space.dt*next->Rhozm)*grid->space.ds);

    const double Coef_hzm = -(2*grid->space.dt)
                            /(2*next->Muz+ grid->space.dt*next->Rhozm);

    next->Hz = Coef_hhz*old[xy].Hz
               + Coef_hex*(old[uy].Ex-old[xy].Ex) 
               - Coef_hex*(old[ux].Ey-old[xy].Ey) 
               + Coef_hzm*old[xy].Mz;
}

__global__ void cuda_update(THGrid * grid)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int x = INDEX_TO_X(i,grid->space.nx);
    int y = INDEX_TO_Y(i,grid->space.nx);

    if(!grid->current)
    {
        updateE(grid,x,y);
    }
    else
    {
        updateH(grid,x,y);
    }  
}

void update(THGrid * grid)
{
    int size = grid->space.nx* grid->space.ny;
    THGrid * cuda_grid;
    
    hipMalloc((void **)&cuda_grid,sizeof(THGrid));
    hipMemcpy(cuda_grid,grid,sizeof(THGrid),hipMemcpyHostToDevice);
    cuda_update<<<size/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(cuda_grid);
    
    grid->time += 0.5*grid->space.dt;
    grid->current = 1 - grid->current;
    hipFree(cuda_grid);

    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout <<  hipGetErrorString(error) << std::endl;
    }
}
