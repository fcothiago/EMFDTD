#include "hip/hip_runtime.h"
#include <TH.h>
#include <utils.h>
#include <iostream>

THGrid::THGrid(unsigned int nx, unsigned int ny, double ds):Grid2D(nx,ny,ds)
{
    THCell * array = new THCell[nx*ny];

    hipError_t err1 = hipMalloc((void **)&this->cells[0],sizeof(THCell)*nx*ny);
    hipError_t err2 = hipMalloc((void **)&this->cells[1],sizeof(THCell)*nx*ny);

    if(err1 != hipSuccess)
        std::cout << hipGetErrorString(err1) << std::endl;
    if(err2 != hipSuccess)
        std::cout << hipGetErrorString(err2) << std::endl;

    err1 = hipMemcpy(this->cells[0],array,sizeof(THCell)*nx*ny,hipMemcpyHostToDevice);
    err2 = hipMemcpy(this->cells[1],array,sizeof(THCell)*nx*ny,hipMemcpyHostToDevice);

    if(err1 != hipSuccess)
        std::cout << hipGetErrorString(err1) << std::endl;
    if(err2 != hipSuccess)
        std::cout << hipGetErrorString(err2) << std::endl;

    delete array;
}

THGrid::~THGrid()
{
    hipFree((void **)&this->cells[0]);
    hipFree((void **)&this->cells[1]);
}

THCell::THCell()
{
    this->Ex = 0;
    this->Ey = 0;
    this->Hz = 0;
    this->Epsx = eps_0;
    this->Epsy = eps_0;
    this->Muz = mu_0;
    this->Rhoxe = 0;
    this->Rhoye = 0;
    this->Rhozm = 0;
}
 
__device__ void updateE(THGrid * grid,int & x,int & y)
{   
    const unsigned int lx = XY_TO_INDEX(x-1,y,grid->space.nx,grid->space.ny);
    const unsigned int ly = XY_TO_INDEX(x,y-1,grid->space.nx,grid->space.ny);
    const unsigned int xy = XY_TO_INDEX(x,y,grid->space.nx,grid->space.ny);

    THCell * next = & grid->cells[1-grid->current][xy] ;
    THCell * old = grid->cells[grid->current] ;

    /*const double Coef_eex = (2*next->Epsx-grid->space.dt*next->Rhoxe)
                    /(2*next->Epsx+grid->space.dt*next->Rhoxe);
    const double Coef_ehx = (2*grid->space.dt)
                    /((2*next->Epsx+grid->space.dt*next->Rhoxe)*grid->space.ds);
    const double Coef_eix = (-2*grid->space.dt)
                    /(2*next->Epsx+grid->space.dt*next->Rhoxe);

    const double Coef_eey = (2*next->Epsy-grid->space.dt*next->Rhoye)
                    /(2*next->Epsy+grid->space.dt*next->Rhoye);
    const double Coef_ehy = (- 2*grid->space.dt)
                    /((2*next->Epsy+grid->space.dt*next->Rhoye)*grid->space.ds);
    const double Coef_eiy = (-2*grid->space.dt)
                    /(2*next->Epsy+grid->space.dt*next->Rhoye);*/

    const double Coef_eex = 1;
    const double Coef_ehx = (grid->space.dt)/((next->Epsx)*grid->space.ds);
    const double Coef_eix = (-grid->space.dt)/(next->Epsx);

    const double Coef_eey = 1;
    const double Coef_ehy = (-grid->space.dt)/((next->Epsy)*grid->space.ds);
    const double Coef_eiy = (-grid->space.dt)/(next->Epsy);


    next->Ex = Coef_eex*old[xy].Ex
            + (mu_0/eps_0)*Coef_ehx*(old[xy].Hz - old[ly].Hz)
            + (mu_0/eps_0)*Coef_eix*(old[xy].Jx);

    next->Ey = Coef_eey*old[xy].Ey 
            + (mu_0/eps_0)*Coef_ehy*(old[xy].Hz - old[lx].Hz) 
            + (mu_0/eps_0)*Coef_eiy*(old[xy].Jy);
}

__device__ void updateH(THGrid * grid,int & x,int & y)
{
    const long unsigned int ux = XY_TO_INDEX(x+1,y,grid->space.nx,grid->space.ny);
    const long unsigned int uy = XY_TO_INDEX(x,y+1,grid->space.nx,grid->space.ny);
    const long unsigned int xy = XY_TO_INDEX(x,y,grid->space.nx,grid->space.ny);

    THCell * next = & grid->cells[1-grid->current][xy] ;
    THCell * old = grid->cells[grid->current] ;

    /*const double Coef_hhz = (2*next->Muz- grid->space.dt*next->Rhozm)
                            /(2*next->Muz+ grid->space.dt*next->Rhozm);

    const double Coef_hex = (2*grid->space.dt)
                            /((2*next->Muz+ grid->space.dt*next->Rhozm)*grid->space.ds);

    const double Coef_hzm = -(2*grid->space.dt)
                            /(2*next->Muz+ grid->space.dt*next->Rhozm);*/

    const double Coef_hhz = 1;

    const double Coef_hex = (grid->space.dt)/((next->Muz)*grid->space.ds);

    const double Coef_hzm = -(grid->space.dt)/(next->Muz);

    next->Hz = Coef_hhz*old[xy].Hz
               + (eps_0/mu_0)*Coef_hex*(old[uy].Ex-old[xy].Ex) 
               - (eps_0/mu_0)*Coef_hex*(old[ux].Ey-old[xy].Ey) 
               + (eps_0/mu_0)*Coef_hzm*old[xy].Mz;
}

__global__  void update(THGrid * grid)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int x = INDEX_TO_X(i,grid->space.nx);
    int y = INDEX_TO_Y(i,grid->space.nx);

    if(!grid->current)
    {
        updateE(grid,x,y);
    }
    else
    {
        updateH(grid,x,y);
    }  
}
