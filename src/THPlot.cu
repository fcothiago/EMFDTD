#include <plot2D.h>
#include <matplotlibcpp.h>

namespace plt = matplotlibcpp;

THPlot::THPlot(THGrid * grid)
{
    this->grid = grid;
    this->cells = std::vector<THCell *>();
}

THPlot::~THPlot()
{
    for(THCell * i : this->cells)
        delete i;
}

void THPlot::capture()
{
    int size = this->grid->space.nx*this->grid->space.ny;
    THCell * print = new THCell[size];
    THCell * current = this->grid->cells[this->grid->current];
    hipError_t err = hipMemcpy(print,current,size*sizeof(THCell),hipMemcpyDeviceToHost);
    if(err != hipSuccess)
        std::cout << hipGetErrorString(err) << std::endl;
    this->cells.push_back(print);
}

void THPlot::plotHz(std::string path)
{ 
    int nx = this->grid->space.nx;
    int ny = this->grid->space.ny;
    float * data = new float[nx*ny];
    int t = 0;
    for(THCell * i : this->cells)
    {
        for(int j = 0; j < nx*ny ; j++)
            data[j] = i[j].Hz;
        plt::imshow(data,ny,nx,1);
        plt::save(path+"/Hz"+std::to_string(t++)+".png");
    }
    delete data;
}