#include <plot2D.h>
#include <utils.h>
#include <matplotlibcpp.h>

#include <iostream>

#include <stdio.h>

namespace plt = matplotlibcpp;

THPlot::THPlot(int n, THGrid * grid)
{
    this->capture_counter = 0;
    this->capture_limit = n;

    this->nx = grid->space.nx;
    this->ny = grid->space.ny;

    this->prints = new THCell * [n];

    for(int i = 0; i < n; i++)
        this->prints[i] = new THCell[this->nx*this->ny];
}

THPlot::~THPlot()
{
    for(int i = 0; i < this->capture_limit; i++)
        delete this->prints[i];
    delete this->prints;
}

void capture(THGrid * grid, THPlot * plot)
{

    if(plot->capture_counter >= plot->capture_limit)
        return;

    THGrid * copy = (THGrid *) malloc(sizeof(THGrid));
    hipMemcpy(copy,grid,sizeof(THGrid),hipMemcpyDeviceToHost);
    hipError_t error = hipMemcpy(plot->prints[plot->capture_counter],copy->cells[copy->current],sizeof(THCell)*plot->nx*plot->ny,hipMemcpyDeviceToHost);
    plot->capture_counter++;

    free(copy);
}

void THPlot::plotHz(std::string path)
{ 
    float * data = new float [this->nx*this->ny];
    for(int i = 0; i < this->capture_limit; i++)
    {
        for(int j = 0; j < this->nx*this->ny; j++)
            data[j] = this->prints[i][j].Hz;

        plt::imshow(data,this->ny,this->nx,1);
        plt::save(path+"/Hz"+std::to_string(i)+".png");

    }

    delete data;
}