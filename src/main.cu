#include "hip/hip_runtime.h"
#include <iostream>
#include <TH.h>
#include <plot2D.h>
#include <string>
#include <utils.h>

#define THREADS_PER_BLOCK 100

__global__ void step(THGrid * grid)
{
    grid->time += 0.5*grid->space.dt;
    grid->current = 1 - grid->current;
}

int main(int argc,char * argv[])
{
    int x = 50;
    int y = 50;
    int nx = 100;
    int ny = 100;
    int size = nx*ny;
    
    THGrid * grid = new THGrid(nx,ny,1);

    THGrid * device_grid = NULL;
    THSinHz * device_wave = NULL;
    THPlot * device_plot = NULL;

    THPlot * plot = new THPlot(200,grid);
    THSinHz * wave = new THSinHz(device_grid,x,y,10.0,6.0E7,0);

    hipMalloc((void **)&device_grid,sizeof(THGrid));
    hipMalloc((void **)&device_wave,sizeof(THSinHz));
    hipMalloc((void **)&device_plot,sizeof(THPlot));

    hipMemcpy(device_grid,grid,sizeof(THGrid),hipMemcpyHostToDevice);
    hipMemcpy(device_wave,wave,sizeof(THSinHz),hipMemcpyHostToDevice);
    hipMemcpy(device_plot,plot,sizeof(THPlot),hipMemcpyHostToDevice);

    for(int i = 0 ; i <= 1000 ;i++)
    {
        update<<<size/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(device_grid);
        step<<<1,1>>>(device_grid);
        update<<<1,1>>>(device_grid,device_wave);
        if((i%5) == 0)
            capture(device_grid,plot);
    }

    plot->plotHz("out/");

    return 0;
}